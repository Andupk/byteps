// Copyright 2020 Bytedance Inc. or its affiliates. All Rights Reserved.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.
// =============================================================================

#include "sparse.h"

namespace byteps {
namespace sparse {

/**
  embedBuffers: the addresses of all embedding buffers (could have variable length)
  denseBuffers: the addresses of all dense buffers (the length should be identical)
  embedBufferLens: the length of the embedding buffers (could have variable length)
  size: the length of a dense buffer (in bytes), it is equivalent for all GPUs
 */
void bytepsSparseInit(std::vector<void*>& embedBuffers, 
                      std::vector<void*>& denseBuffers, 
                      std::vector<int>& embedBufferLens, 
                      int size) {
  BytePSSparseCommon::Init();
  CHECK_EQ(embedBuffers.size(), denseBuffers.size());
  CHECK_EQ(embedBufferLens.size(), denseBuffers.size());
  
  // Init IPC stuff
  sharedMemoryInfo info;
  CHECK_EQ(sharedMemoryCreate(bpsShmName, sizeof(shmStruct), &info), 0);
  auto shm = (volatile shmStruct *)info.addr;
  memset((void *)shm, 0, sizeof(*shm));

  auto localSize = BytePSSparseCommon::GetLocalSize();
  auto workerNum = BytePSSparseCommon::GetNumWorker();
  auto workerID = BytePSSparseCommon::GetWorkerID();

  for (int i = 0; i < localSize; i++) {
    hipDeviceProp_t prop;
    CUDA_CALL(hipGetDeviceProperties(&prop, i));

    // CUDA IPC is only supported on devices with unified addressing
    if (!prop.unifiedAddressing) {
      // BPS_LOG(INFO) << "Device " << i << " does not support unified addressing, skipping...";
      continue;
    }
    // We require two processes accessing each device, so we need
    // to ensure exclusive or prohibited mode is not set
    if (prop.computeMode != hipComputeModeDefault) {
      // BPS_LOG(INFO) << "Device " << i << "is in an unsupported compute mode for this sample";
      continue;
    }

    shm->devices[shm->nprocesses++] = i;
    CHECK_GT(MAX_CUDA_DEVICES, shm->nprocesses);
  }
  CHECK(shm->nprocesses > 0) 
      << "No cuda device suppported";
  CHECK_EQ(shm->nprocesses, embedBuffers.size())
      << "Shared memory processes: " << shm->nprocesses 
      << ", send buffers: " << embedBuffers.size();

  // We need to manually we need to clear the containers because
  // bytepsSparseInit() might be (unexpectedly) invoked multiple times
  _embedBuffers.assign(embedBuffers.begin(), embedBuffers.end());
  _denseBuffers.assign(denseBuffers.begin(), denseBuffers.end());

  _embedBufferLens.clear();
  _embedBufferLens.resize(workerNum);
  for (int i = 0; i < workerNum; i++) {
    _embedBufferLens[i].resize(localSize);
  }

  // Allocate memory and an event for each process and fill 
  // the shared memory buffer with the IPC handles 
  for (size_t i = 0; i < shm->nprocesses; i++) {
    hipEvent_t event;
    CUDA_CALL(hipSetDevice(
        shm->devices[i]));
    CUDA_CALL(hipIpcGetMemHandle(
        (hipIpcMemHandle_t *)&shm->embedMemHandle[i], embedBuffers[i]));
    CUDA_CALL(hipIpcGetMemHandle(
        (hipIpcMemHandle_t *)&shm->denseMemHandle[i], denseBuffers[i]));
    CUDA_CALL(hipEventCreate(
        &event, hipEventDisableTiming | hipEventInterprocess));
    CUDA_CALL(hipIpcGetEventHandle(
        (hipIpcEventHandle_t *)&shm->eventHandle[i], event));
    
    // Store the buffers 
    _embedBufferLens[workerID][i] = embedBufferLens[i]; // local buffer length
  }
  _denseBufferLen = size;

  // Check buffer length
  int accuml = 0;
  for (int i = 0; i < localSize; i++) {
    accuml += _embedBufferLens[workerID][i] / localSize;
  }
  CHECK_EQ(accuml, _denseBufferLen) 
      << accuml << " " << _denseBufferLen;

  // Need a continous CPU buffer for each GPU
  _cpuBuffers.clear();
  for (int i = 0; i < localSize; i++) {
    void* _cpuBuffer;
    CUDA_CALL(hipHostAlloc(
        &_cpuBuffer, size, hipHostMallocMapped | hipHostMallocPortable));
    _cpuBuffers.push_back(_cpuBuffer);
  }
  
  // The followings are for the global coordination of 
  // the embedding buffer length, which is equivalent to all-gather 
  auto ps = BytePSSparseCommon::GetPS();
  if (BytePSSparseCommon::IsDistributed()) {
    CHECK(ps); // must init the pslite instance before
    std::vector<ps::SArray<char>> bufferLenSarrays;
    for (int i = 0; i < workerNum; i++) {
      ps::SArray<char> tmp(
          (char*)_embedBufferLens[i].data(), localSize * sizeof(int), false);
      bufferLenSarrays.push_back(tmp);
    }
    std::vector<ps::SArray<ps::Key>> tmpKeys;
    std::vector<ps::SArray<int>> tmpLens;
    auto krs = ps::Postoffice::Get()->GetServerKeyRanges();
    for (int key = 0; key < workerNum; key++) {
      int server = key;

      std::vector<ps::Key> tmp1(1, krs[server].begin() + key);
      ps::SArray<ps::Key> keys(tmp1);
      tmpKeys.push_back(keys);

      std::vector<int> tmp2(1, localSize * sizeof(int));
      ps::SArray<int> lens(tmp2);
      tmpLens.push_back(lens);
    }

    // Push once to the associated server
    {
      int server = workerID;
      auto keys = tmpKeys[server];
      auto vals = bufferLenSarrays[server];
      auto lens = tmpLens[server];
      ps->Wait(ps->ZPush(keys, vals, lens));
    }

    // Call a barrier to sync across multiple workers. 
    // In case that some workers finish push too fast, 
    // and then pull from other workers too early
    ps::Postoffice::Get()->Barrier(
        0, ps::kWorkerGroup + ps::kServerGroup + ps::kScheduler);

    // Gather from other workers 
    for (int key = 0; key < workerNum; key++) {
      int server = key;
      if (server == workerID) continue; // skip myself
      auto keys = tmpKeys[server];
      auto vals = bufferLenSarrays[server];
      auto lens = tmpLens[server];
      ps->Wait(ps->ZPull(keys, &vals, &lens));
    }
  }

  // Prepare gossip-gather communication
  _local_gather_comms.resize(localSize);
  for (int i = 0; i < localSize; i++) {
    std::vector<float*> srcs(localSize);
    std::vector<size_t> srcs_lens(localSize);
    std::vector<size_t> send_counts(localSize);
    for (int j = 0; j < localSize; j++) {
      srcs[j] = (float*)_embedBuffers[j] + (i * _embedBufferLens[workerID][j] / localSize);
      srcs_lens[j] = (localSize - i) * _embedBufferLens[workerID][j] / localSize;
      send_counts[j] = _embedBufferLens[workerID][j] / localSize;
    }
    float* dst = (float *)_denseBuffers[i];
    size_t dst_len = _denseBufferLen;

    std::string planfile_name("gather_plan_");
    planfile_name += std::to_string(i) + std::string(".json");
    _local_gather_comms[i] = std::make_unique<LocalGatherComm>(
        planfile_name, localSize, srcs, srcs_lens, send_counts, dst, dst_len);
  }

  // Prepare gossip-scatter communication
  _local_scatter_comms.resize(localSize);
  for (int i = 0; i < localSize; i++) {
    float* src = (float *)_denseBuffers[i];
    size_t src_len = _denseBufferLen;
    std::vector<float*> scatter_dsts(localSize);
    std::vector<size_t> scatter_dsts_lens(localSize);
    std::vector<size_t> scatter_send_counts(localSize);
    for (int j = 0; j < localSize; j++) {
      scatter_dsts[j] = (float*)_embedBuffers[j] + (i * _embedBufferLens[workerID][j] / localSize);
      scatter_dsts_lens[j] = (localSize - i) * _embedBufferLens[workerID][j] / localSize;
      scatter_send_counts[j] = _embedBufferLens[workerID][j] / localSize;
    }

    std::string planfile_name("scatter_plan_");
    planfile_name += std::to_string(i) + std::string(".json");
    _local_scatter_comms[i] = std::make_unique<LocalScatterComm>(
        planfile_name, localSize, src, src_len, scatter_send_counts, scatter_dsts, scatter_dsts_lens);
  }

} 

void bytepsSparseShutdown() {
}


void bytepsGatherExecAsync(int local_rank, hipStream_t stream) {
  // Gather from local peer GPUs on the same worker
  auto localSize = BytePSSparseCommon::GetLocalSize();
  auto workerID = BytePSSparseCommon::GetWorkerID();
  auto workerNum = BytePSSparseCommon::GetNumWorker();

  _local_gather_comms[local_rank]->ExecAsync();
}

void bytepsSynchronize(int local_rank, hipStream_t stream, OP op) { 
  switch (op) {
    case GATHER:
      _local_gather_comms[local_rank]->Sync();
      break;
    case SCATTER:
      _local_scatter_comms[local_rank]->Sync();
      break;
    default:
      CHECK(0) << "unrecognized operation";
  }
  CUDA_CALL(hipStreamSynchronize(stream));
}

void bytepsScatterExecAsync(int local_rank, hipStream_t stream) {
  auto localSize = BytePSSparseCommon::GetLocalSize();
  auto workerID = BytePSSparseCommon::GetWorkerID();
  void* baseSrcPtr = (void*)_denseBuffers[local_rank];

  _local_scatter_comms[local_rank]->ExecAsync();
}


} // namespace sparse
} // namespace byteps 